#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "mnist_model.h"

__global__ void kernel_madd(float* A, float* B, float* C, int M, int N);
__global__ void kernel_mmelem(float* A, float* B, float* C, int M, int N);
__global__ void kernel_mmreduce(float* A, float* B, int M, int N);
__global__ void kernel_mm(float* A, float* B, float* C, int N_a, int M_a, int M_b);
__global__ void kernel_mt(float* A, float* B, int M, int N);


void cu_madd(float* A, float* B, float* C, int M, int N)
{
	float *d_a, *d_b, *d_c;

	dim3 blk;
	blk.x = 16; blk.y = 16;

	dim3 grid;
	grid.x = (M + blk.x - 1) / blk.x;
	grid.y = (N + blk.y - 1) / blk.y;

	int size = sizeof(float)*M*N;

	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);

	hipMemcpy(d_a, A, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, B, size, hipMemcpyHostToDevice);

	kernel_madd << < grid, blk >> > (d_a, d_b, d_c, M, N);

	hipMemcpy(C, d_c, size, hipMemcpyDeviceToHost);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
}

void cu_mmelem(float* A, float* B, float* C, int M, int N)
{
	float *d_a, *d_b, *d_c;

	dim3 blk;
	blk.x = 16; blk.y = 16;

	dim3 grid;
	grid.x = (M + blk.x - 1) / blk.x;
	grid.y = (N + blk.y - 1) / blk.y;

	int size = sizeof(float)*M*N;

	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);

	hipMemcpy(d_a, A, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, B, size, hipMemcpyHostToDevice);

	kernel_mmelem << < grid, blk >> > (d_a, d_b, d_c, M, N);

	hipMemcpy(C, d_c, size, hipMemcpyDeviceToHost);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
}

void cu_mmreduce(float* A, float* B, int M, int N)
{
	float *d_a, *d_b;

	int blk = 256;
    int grid = (M + blk - 1) / blk;
	int sizeA = sizeof(float)*M*N;
    int sizeB = sizeof(float)*M;

	hipMalloc((void **)&d_a, sizeA);
	hipMalloc((void **)&d_b, sizeB);

	hipMemcpy(d_a, A, sizeA, hipMemcpyHostToDevice);

	kernel_mmreduce << < grid, blk >> > (d_a, d_b, M, N);

	hipMemcpy(A, d_b, sizeB, hipMemcpyDeviceToHost);

	hipFree(d_a);
	hipFree(d_b);
}

void cu_mm(float* A, float* B, float* C, int N_a, int M_a, int M_b)
{
	float *d_a, *d_b, *d_c;

	dim3 blk;
	blk.x = 16; blk.y = 16;

	dim3 grid;
	grid.x = (M_b + blk.x - 1) / blk.x;
	grid.y = (N_a + blk.y - 1) / blk.y;

	int sizeA = sizeof(float)*M_a*N_a;
    int sizeB = sizeof(float)*M_b*M_a;
	int sizeC = sizeof(float)*N_a*M_b;

	hipMalloc((void **)&d_a, sizeA);
	hipMalloc((void **)&d_b, sizeB);
	hipMalloc((void **)&d_c, sizeC);

	hipMemcpy(d_a, A, sizeA, hipMemcpyHostToDevice);
	hipMemcpy(d_b, B, sizeB, hipMemcpyHostToDevice);

	kernel_mm << < grid, blk >> > (d_a, d_b, d_c, N_a, M_a, M_b);

	hipMemcpy(C, d_c, sizeC, hipMemcpyDeviceToHost);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
}

void cu_mt(float* A, float* B, int M, int N)
{
	float *d_a, *d_b;

	dim3 blk;
	blk.x = 16; blk.y = 16;

	dim3 grid;
	grid.x = (M + blk.x - 1) / blk.x;
	grid.y = (N + blk.y - 1) / blk.y;

	int size = sizeof(float)*M*N;

	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	
	hipMemcpy(d_a, A, size, hipMemcpyHostToDevice);

	kernel_mt << < grid, blk >> > (d_a, d_b, M, N);

	hipMemcpy(B, d_b, size, hipMemcpyDeviceToHost);

	hipFree(d_a);
	hipFree(d_b);
}

__global__ void kernel_madd(float* A, float* B, float* C, int M, int N)
{
	unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
	unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;
	unsigned int idx = iy * M + ix;

	if (ix < M && iy < N)
		C[idx] = A[idx] + B[idx];
}

__global__ void kernel_mmelem(float* A, float* B, float* C, int M, int N)
{
	unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
	unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;
	unsigned int idx = iy * M + ix;

	if (ix < M && iy < N)
		C[idx] = A[idx] * B[idx];
}

__global__ void kernel_mmreduce(float* A, float* B, int M, int N)
{
	unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;

	float sum = 0.0f;
	for (int i = 0; i < N; i++) {
        sum += A[i * M + ix];
	}
	B[ix] = sum;
}

__global__ void kernel_mm(float *A, float *B, float *C, int N_a, int M_a, int M_b) 
{
	unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int iy = blockIdx.y * blockDim.y + threadIdx.y;

	if (iy >= N_a || ix >= M_b)
		return;

	float sum = 0.0f;
	for (int i = 0; i < M_a; i++)
		sum += A[iy * M_a + i] * B[i * M_b + ix];
	C[iy * M_b + ix] = sum;
}

__global__ void kernel_mt(float* A, float* B, int M, int N)
{
	__shared__ float tile[16][16];

	// Coordinates of original matrix
	unsigned int ix, iy, ti, to;
	ix = threadIdx.x + blockIdx.x * blockDim.x;
	iy = threadIdx.y + blockIdx.y * blockDim.y;
	ti = iy * M + ix;
	
	if (iy < N && ix < M)
	{
		tile[threadIdx.y][threadIdx.x] = A[ti]; // load to shared memory
	}
	
	unsigned int bidx, irow, icol;
	bidx = threadIdx.y * blockDim.x + threadIdx.x;
	irow = bidx / blockDim.y;
	icol = bidx % blockDim.y;
	// Coordinates of transposed matrix
	ix = blockIdx.y * blockDim.y + icol;
	iy = blockIdx.x * blockDim.x + irow;
	to = iy * N + ix;
	__syncthreads();

	if (iy < M && ix < N) {
		B[to] = tile[icol][irow]; // load back to global memory
	}
}